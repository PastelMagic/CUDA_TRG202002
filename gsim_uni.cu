//-------------------------------------------------
//- GPU演算テスト（ユニファイドメモリ版）
//-　　物体間、及び中心位置から距離の自乗に比例する
//-    ような吸引力が働くとして動かしてみた
//- コンパイル時のオプション
//- nvcc -o xxx xxx.cu -lX11 -lm
//- CPUと同じコードをGPUを１スレッドで動かしたい時は
//-   -D USE_GPU_SINGLEを付加
//- CPUで動かしたい時は
//-   -D USE_CPUを付加
//- ex.
//- nvcc -o xxx xxx.cu -D USE_CPU -lX11 -lm
//-------------------------------------------------
//

// #define USE_CPU
// #define USE_GPU_SINGLE

#include <stdio.h>
#include <math.h>
#include <X11/Xlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>


#define WIN_W     600	// ウインドウの横幅
#define WIN_H     600	// ウインドウの縦幅
#define CENT_X    (WIN_W>>1)	// X方向の中央位置
#define CENT_Y    (WIN_H>>1)	// Y方向の中央位置
#define WIN_X     100	// ウインドウの左上のX座標
#define WIN_Y     100	// ウインドウの左上のY座法
#define BORD_W    2	// ボーダ幅

#define	OBJN	10	// ボールの数（2^N個)
#define	OBJS	(1<<OBJN)	// ボールの数（N個）

unsigned long  bcol;
unsigned long  black,white; /* 黒と白のピクセル値 */


#ifndef USE_CPU
__device__ __managed__ double *posX, *posY;	//pos[2][OBJS];
__device__ __managed__ double *spdX, *spdY;	//spd[2][OBJS];
__device__ __managed__ double wt[OBJS];		//wt[OBJS];
__device__ __managed__ int Nobj;	// ユニファイドメモリ上の変数として定義

#else
double *posX, *posY;	//pos[2][OBJS];
double *spdX, *spdY;	//spd[2][OBJS];
double wt[OBJS];	//wt[OBJS];
int Nobj;

#endif


#ifndef USE_CPU
__global__ void GPUmove()
{
	int Start, End, Size;
	int NThreads;
	int thread;

	NThreads = gridDim.x * blockDim.x;	// 総スレッド数を算出
	if (NThreads >= Nobj)			// データ数よりスレッド数の方が多い
		NThreads = Nobj;		// データ数に合わせる
	Size = Nobj / NThreads;			// 1スレッド当たりの担当数（割り切れる前提）
	thread = blockDim.x * blockIdx.x + threadIdx.x;	// 自分のスレッド番号
	if (thread < NThreads) {		// スレッド数未満なら担当分あり
		Start = thread * Size;
		End = Start + Size;
		for (int i=Start; i<End; i++) {
			double fx,fy;
			// 誤差蓄積などで早期に飛び散っていかないように
			// 中心に寄せる力を加えておく
			fx= -posX[i];
			fx *= fx*fx/512;
			fy= -posY[i];
			fy *= fy*fy/512;
	
			int targ;		//　引き合い力を計算する相手
			for (int j=1; j<Nobj; j++) {		// 自分との関係は計算しなくていい
				targ = (i+j) & (Nobj-1);	// 相手を決めて
				double dfX = posX[targ] - posX[i];	// X方向の距離
				double dfY = posY[targ] - posY[i];	// Y方向の距離
				double dist = sqrt(dfX*dfX + dfY*dfY);	// 直線距離
				fx += dfX*dist/1024;	// 距離の自乗に比例したX方向の力
				fy += dfY*dist/1024;	// 距離の自乗に比例したY方向の力
			}
			spdX[i] += (fx/wt[i]);		// X方向の加速度に応じて速度を加算
			spdY[i] += (fy/wt[i]);		// Y方向の加速度に応じて速度を加算
		}
		for (int i=Start; i<End; i++) {		// すべての速度を算出したので
			posX[i] += spdX[i];		// X方向の速度分だけ座標移動
			posY[i] += spdY[i];		// Y方向の速度分だけ座標移動
		}
	}
}
#endif

#ifdef USE_GPU_SINGLE
__global__ void move()
#else
void move()
#endif
{
	for (int i=0; i<Nobj; i++) {
		double fx,fy;
		// 誤差蓄積などで早期に飛び散っていかないように
		// 中心に寄せる力を加えておく
		fx= -posX[i];
		fx *= fx*fx/512;
		fy= -posY[i];
		fy *= fy*fy/512;

		int targ;		//　引き合い力を計算する相手
		for (int j=1; j<Nobj; j++) {		// 自分との関係は計算しなくていい
			targ = (i+j) & (Nobj-1);	// 相手を決めて
			double dfX = posX[targ] - posX[i];	// X方向の距離
			double dfY = posY[targ] - posY[i];	// Y方向の距離
			double dist = sqrt(dfX*dfX + dfY*dfY);	// 直線距離
			fx += dfX*dist/1024;	// 距離の自乗に比例したX方向の力
			fy += dfY*dist/1024;	// 距離の自乗に比例したY方向の力
		}
		spdX[i] += (fx/wt[i]);		// X方向の加速度に応じて速度を加算
		spdY[i] += (fy/wt[i]);		// Y方向の加速度に応じて速度を加算
	}
	for (int i=0; i<Nobj; i++) {		// すべての速度を算出したので
		posX[i] += spdX[i];		// X方向の速度分だけ座標移動
		posY[i] += spdY[i];		// Y方向の速度分だけ座標移動
	}
}


//-----------------------------------
//-- 移動処理呼び出しディスパッチャ
//-----------------------------------
//
int DataBytes = sizeof(double) * OBJS;		// 配列データの総バイト数を計算
						// 全部Doubleなので共通で使う
void DispatchMove()
{
#ifndef USE_CPU
  #ifndef USE_GPU_SINGLE	// シングルスレッドじゃない（マルチスレッド）
	GPUmove<<<32, 32>>>();
	hipDeviceSynchronize();	// GPUの演算終了を待つ
  #else				// シングルスレッド
	move<<<1, 1>>>();
//	GPUmove<<<1, 1>>>(posX, posY, spdX, spdY, wt);
	cudaDeviceSynchronize();	// GPUの演算終了を待つ
  #endif
#else				// CPUで頑張る
	move();
#endif
}


//-----------------------------------
//-- 配列等の初期化
//-----------------------------------
//
void init(void)
{
	Nobj = OBJS;
#ifndef USE_CPU
	hipMallocManaged(&posX, DataBytes);	// ユニファイドメモリ確保
	hipMallocManaged(&posY, DataBytes);
	hipMallocManaged(&spdX, DataBytes);
	hipMallocManaged(&spdY, DataBytes);
#else
	posX = (double *)malloc(DataBytes);	// CPU使用時はmalloc()で済ませる
	posY = (double *)malloc(DataBytes);
	spdX = (double *)malloc(DataBytes);
	spdY = (double *)malloc(DataBytes);
#endif
	for (int i=0; i<OBJS; i++) {
		spdX[i] = 0;		// 速度のX方向初期値はゼロ
		spdY[i] = 0;;		// 速度のY方向初期値はゼロ
		wt[i] = OBJS*1000+(i & ~0x3)*800;	// 4個おきに重さを変化させてバランスさせる
							// 数が増えると掛かる力も増加しがちなので、
							// 重さも増えるようにした
	}
	// オブジェクトの初期位置（結構適当にやってる）
	double p = (double)WIN_W/(double)OBJS;
	double stp = p;
	double offset = WIN_W/64/OBJS;
	for (int i=0; i<OBJS; i+= 4, p+=stp) {
		offset = i*i*WIN_W/OBJS/OBJS/8;
		posX[i] = p;
		posY[i] = offset;

		posX[i+1]= -offset;
		posY[i+1] = p;

		posX[i+2] = -p;
		posY[i+2] = -offset;

		posX[i+3] = offset;
		posY[i+3] = -p;
	}
}


//-----------------------------------
//-- GPU上に確保したメモリ領域解放
//-----------------------------------
void release()
{
#ifndef USE_CPU
	hipFree(posX);
	hipFree(posY);
	hipFree(spdX);
	hipFree(spdY);
#else
	free(posX);
	free(posY);
	free(spdX);
	free(spdY);
#endif
}

//-----------------------------------
//-- 指定位置に四角を描く
//-----------------------------------
//
static void draw_dot(Display *dpy, Window win, GC gc,
  unsigned int x, unsigned int y, unsigned long color)
{
	XSetForeground( dpy, gc, color );
//	XFillArc( dpy, win, gc,x, y, 5, 5, 0, 360*64);
	XFillRectangle(dpy, win, gc, x-1, y-1, 2, 2);
}

//-----------------------------------
//-- オブジェクトの描画
//-----------------------------------
//
void draw(Display *dpy, Window win, GC gc)
{
	XSetForeground( dpy, gc, black );
	XFillRectangle(dpy, win, gc, 0, 0, WIN_W, WIN_H);
	for (int i=0; i<OBJS; i++) {
		int x = (int)posX[i]+CENT_X;
		int y = (int)posY[i]+CENT_Y;
		draw_dot(dpy, win, gc, x, y, bcol);
	}
}

int main( void )
{
	Display*       dpy;         /* ディスプレイ */
	Window         root;        /* ルートウィンドウ */
	Window         win;         /* 表示するウィンドウ */
	int            screen;      /* スクリーン */
	GC             gc;          /* グラフィックスコンテキスト */
	XEvent         evt;         /* イベント構造体 */
	Colormap       cmap;        /* カラーマップ */
	XColor         color, exact;

	init();

	/* Xサーバと接続する */
	dpy = XOpenDisplay( "" );

	/* ディスプレイ変数の取得 */
	root   = DefaultRootWindow( dpy );
	screen = DefaultScreen( dpy );
	/* XAllocNamedColor() のためにカラーマップを取得 */
	cmap   = DefaultColormap( dpy, screen );

	white  = WhitePixel( dpy, screen );
	black  = BlackPixel( dpy, screen );
	XAllocNamedColor( dpy, cmap, "MistyRose", &color, &exact );
	bcol = color.pixel;

	// ウインドウの作成
	win = XCreateSimpleWindow( dpy, root,
	   WIN_X, WIN_Y, WIN_W, WIN_H, BORD_W, white, black);
	// グラフィックコンテキスト作成
	gc = XCreateGC( dpy, win, 0, NULL );
	// 再描画と、キーボード押下（終了させるのに使っている）イベントを取得
	XSelectInput( dpy, win, KeyPressMask | ExposureMask );
	// ウインドウ表示
	XMapWindow( dpy, win );

	int LoopCount = 0;
	Bool terminate = False;		// プログラム終了フラグ
	while( !terminate) {
		if (XEventsQueued(dpy, QueuedAfterFlush) != 0) {	// イベントが入っている
			XNextEvent( dpy, &evt );	// イベントをとり出す
			switch( evt.type ) {
				case Expose:		// 再描画が必要になった
					if ( evt.xexpose.count == 0 ) {
						draw( dpy, win, gc);
					}
					break;
				case KeyPress:		// キーが押されたら
					XFreeGC( dpy, gc );	// コンテキストを開放して
					XDestroyWindow( dpy, win );	// ウインドウを破棄
					XCloseDisplay( dpy );		// 表示終了
					terminate = True;		// 終了する
					break;
				default:
					break;
			}
		}
		if (!terminate) {
			DispatchMove();		// 移動処理を呼び出す
			draw(dpy, win, gc);	// 描画
//			usleep(10*1000);	// 速すぎる時にちょっと待たせたいならここ
			LoopCount++;
		}
	}
	printf("Loop:-%d\n",LoopCount);
	release();
	return 0;
}
